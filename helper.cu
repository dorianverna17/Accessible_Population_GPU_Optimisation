#include "hip/hip_runtime.h"
#include <math.h>
#include <fstream>
#include <iostream>

#include "helper.h"

using namespace std;

#define MAX_ELEM 1000000

/*
 * geoDistance_kernel computes geographical distance (lat1, lat1) and (lat2, lon2)
 * It is the kernel function that gets called
 */
__global__ void geoDistance_kernel(float *lat, float *lon, const size_t size, int *pop, int *res, float kmRange)
{
    /* get the index of the city for which we compute the accessible population */
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    /* if index exceeds the size of the array, then skip computation */
    if (i >= size) {
            return;
    }

    /* initialize vector with 0 */
    res[i] = 0;

    /* loop through all the cities, first initialize some data needed */
    float register lat1 = (90 - lat[i]) * DEGREE_TO_RADIANS;
    float register lon1 = lon[i] * DEGREE_TO_RADIANS;
    float register radius = 6371 * 6371;
    float register kmRange_pythagoras = kmRange * kmRange;
    float register lat2, lon2, distance_x, distance_y, distance;
    for (int j = 0; j < size; j += 3) {
        /* take the latitude */
        lat2 = (90 - lat[j]) * DEGREE_TO_RADIANS;

        /* take the longitude */
        lon2 = lon[j] * DEGREE_TO_RADIANS;

        /* use Pythagora to compute the two sides of the triangle */
        distance_x = (lat2 - lat1) * (lat2 - lat1);
        distance_y = (lon2 - lon1) * (lon2 - lon1);

        /* get the desired side of the triangle */
        distance = distance_x + distance_y;

        /* check kmRange with the actual distance */
        if (distance * radius <= kmRange_pythagoras)
                res[i] += pop[j];

        /* perform second step of loop unrolling */
        lat2 = (90 - lat[j + 1]) * DEGREE_TO_RADIANS;
        lon2 = lon[j + 1] * DEGREE_TO_RADIANS;
        distance_x = (lat2 - lat1) * (lat2 - lat1);
        distance_y = (lon2 - lon1) * (lon2 - lon1);
        distance = distance_x + distance_y;
        if (distance * radius <= kmRange_pythagoras)
                res[i] += pop[j + 1];

        /* perform third step of loop unrolling */
        lat2 = (90 - lat[j + 2]) * DEGREE_TO_RADIANS;
        lon2 = lon[j + 2] * DEGREE_TO_RADIANS;
        distance_x = (lat2 - lat1) * (lat2 - lat1);
        distance_y = (lon2 - lon1) * (lon2 - lon1);
        distance = distance_x + distance_y;
        if (distance * radius <= kmRange_pythagoras)
                res[i] += pop[j + 2];
    }
}

// sampleFileIO demos reading test files and writing output
void sampleFileIO(float kmRange, const char* fileIn, const char* fileOut)
{
    /* declare the needed host buffers */
    string register geon;
    float register *lat = (float *) malloc(MAX_ELEM * sizeof(float));
    float register *lon = (float *) malloc(MAX_ELEM * sizeof(float));
    int register *pop = (int *) malloc(MAX_ELEM * sizeof(int));

    int register *res = (int *) malloc(MAX_ELEM * sizeof(int));

    /* declare and initialize file descriptors */
    ifstream ifs(fileIn);
    ofstream ofs(fileOut);

    /* declare and alloc device buffers */
    float register *lat_device = 0;
    float register *lon_device = 0;
    int register *res_device = 0;
    int register *pop_device = 0;

    hipMalloc((void **) &lat_device, MAX_ELEM * sizeof(float));
    hipMalloc((void **) &lon_device, MAX_ELEM * sizeof(float));
    hipMalloc((void **) &res_device, MAX_ELEM * sizeof(int));
    hipMalloc((void **) &pop_device, MAX_ELEM * sizeof(int));

    /* read from file */
    int it = 0;

    while(ifs >> geon >> lat[it] >> lon[it] >> pop[it])
        it++;

    /* copy data to device */
    hipMemcpy(lat_device, lat, MAX_ELEM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(lon_device, lon, MAX_ELEM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(res_device, res, MAX_ELEM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pop_device, pop, MAX_ELEM * sizeof(int), hipMemcpyHostToDevice);

    /* declare block size and number of blocks */
    const size_t register block_size = 256;
    size_t register no_blocks = it / block_size;

    if (it % block_size != 0)
        no_blocks += 1;

    /* call the kernel function */
    geoDistance_kernel<<<no_blocks, block_size>>>(lat_device, lon_device, it, pop_device, res_device, kmRange);

    /* synchronize devices */
    hipDeviceSynchronize();

    /* copy results from device to host */
    hipMemcpy(res, res_device, MAX_ELEM * sizeof(int), hipMemcpyDeviceToHost);

    /* print results to file */
    for (int i = 0; i < it; i++) {
        ofs << res[i] << endl;
    }

    /* free allocated memory */
    hipFree(lat_device);
    hipFree(lon_device);
    hipFree(res_device);
    hipFree(pop_device);
    free(lat);
    free(lon);
    free(res);
    free(pop);
    ifs.close();
    ofs.close();
}